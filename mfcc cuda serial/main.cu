#include "hip/hip_runtime.h"
#define _CRT_SECURE_NO_WARNINGS
#define _USE_MATH_DEFINES

#include "hip/hip_runtime.h"
#include ""

#include <iostream>
#include <fstream>
#include <sstream>

#include <stdlib.h>
#include <string>

#include <chrono>
#include <cmath>

#include "kernel.h"
#include "Wav.h"

constexpr char FILE_PATH[] = "C:\\Users\\nyoon\\Music\\wav\\s1_anger_M_a1.wav";

constexpr float PRE_EMPHASIS = 0.97;
constexpr float FRAME_SIZE = 0.032;
constexpr float FRAME_STRIDE = 0.016;
constexpr int BIT_LENGTH = 9;
constexpr int NFFT = 1 << BIT_LENGTH;
constexpr int NFILTER = 40;
constexpr int NUM_CEPS = 12;
constexpr float CEP_LIFTER = 23.0;

template <typename T>
void PrintArray(int length, T arr)
{
    for (int i = 0; i < length; ++i)
    {
        std::cout << arr[i] << ", ";

        if ((i + 1) % 8 == 0)
        {
            std::cout << std::endl;
        }
    }

    std::cout << std::endl;
}

int main()
{
    hipError_t cudaStatus;

    // load wav file
    Wav wav{ FILE_PATH };

    const int signalLen = wav.GetLen();
    const int sr = 16000;

    short* signal_h;

    cudaStatus = hipHostMalloc((void**)&signal_h, signalLen * sizeof(short));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipHostMalloc failed!: %s\n", hipGetErrorString(cudaStatus));

        return 1;
    }

    wav.GetData(signal_h);

    const int frameLength = FRAME_SIZE * sr;
    const int frameStep = FRAME_STRIDE * sr;
    const int numFrames = ceil(float(signalLen - frameLength) / frameStep) + 1;

    // create events
    hipEvent_t startEvent, stopEvent, dummyEvent;

    cudaStatus = hipEventCreate(&startEvent);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipEventCreate: startEvent, failed!");

        return 1;
    }

    cudaStatus = hipEventCreate(&stopEvent);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipEventCreate: stopEvent, failed!");

        return 1;
    }

    cudaStatus = hipEventCreate(&dummyEvent);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipEventCreate: dummyEvent, failed!");

        return 1;
    }

    //----------------------------------//
    //           Pre-Processing         //
    //----------------------------------//

    short* signal_d;
    float* emphasised_d;

    cudaStatus = hipMalloc((void**)&signal_d, signalLen * sizeof(short));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!: %s\n", hipGetErrorString(cudaStatus));

        return 1;
    }

    cudaStatus = hipMalloc((void**)&emphasised_d, signalLen * sizeof(float));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!: %s\n", hipGetErrorString(cudaStatus));

        return 1;
    }

    //----------------------------------//
    //      FFT and Power Spectrum      //
    //----------------------------------//

    float* powFrames_d;

    cudaStatus = hipMalloc((void**)&powFrames_d, numFrames * (NFFT / 2 + 1) * sizeof(float));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!: %s\n", hipGetErrorString(cudaStatus));

        return 1;
    }

    //----------------------------------//
    //           Filter Banks           //
    //----------------------------------//

    float lowFreqMel = 0;
    float highFreqMel = 2595.0 * log10(1 + (sr / 2.0) / 700.0);
    float melPoints[NFILTER + 2];

    float melStride = (highFreqMel - lowFreqMel) / (NFILTER + 1);
    for (int i = 0; i < NFILTER + 2; ++i)
    {
        melPoints[i] = lowFreqMel + melStride * i;
    }

    float melBins[NFILTER + 2];
    for (int i = 0; i < NFILTER + 2; ++i)
    {
        melBins[i] = floor((NFFT + 1) * (700.0 * (pow(10, (melPoints[i] / 2595.0)) - 1)) / float(sr));
    }

    const int fftResultLen = NFFT / 2 + 1;

    float* fbank = new float[NFILTER * fftResultLen];
    memset(fbank, 0, sizeof(float) * (NFILTER * fftResultLen));

    int nonZeroNum = 0;

    float f_m_minus, f_m, f_m_plus;
    for (int m = 1; m <= NFILTER; ++m)
    {
        f_m_minus = melBins[m - 1];
        f_m = melBins[m];
        f_m_plus = melBins[m + 1];

        for (int k = f_m_minus; k < f_m; ++k)
        {
            fbank[(m - 1) * fftResultLen + k] = (k - f_m_minus) / (f_m - f_m_minus);

            if (fbank[(m - 1) * fftResultLen + k] != 0)
            {
                ++nonZeroNum;
            }
        }

        for (int k = f_m; k < f_m_plus; ++k)
        {
            fbank[(m - 1) * fftResultLen + k] = (f_m_plus - k) / (f_m_plus - f_m);
            
            if (fbank[(m - 1) * fftResultLen + k] != 0)
            {
                ++nonZeroNum;
            }
        }
    }

    float* fbanks_val_h;
    int* fbanks_col_h;
    int* fbanks_row_h;

    hipHostMalloc((void**)&fbanks_val_h, nonZeroNum * sizeof(float));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipHostMalloc failed!: %s\n", hipGetErrorString(cudaStatus));

        return 1;
    }

    hipHostMalloc((void**)&fbanks_col_h, (NFILTER + 2) * sizeof(int));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipHostMalloc failed!: %s\n", hipGetErrorString(cudaStatus));

        return 1;
    }

    hipHostMalloc((void**)&fbanks_row_h, nonZeroNum * sizeof(int));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipHostMalloc failed!: %s\n", hipGetErrorString(cudaStatus));

        return 1;
    }

    fbanks_col_h[0] = 0;
    nonZeroNum = 0;

    for (int i = 0; i < NFILTER; ++i)
    {
        for (int k = 0; k < fftResultLen; ++k)
        {
            if (fbank[i * fftResultLen + k] != 0)
            {
                fbanks_val_h[nonZeroNum] = fbank[i * fftResultLen + k];
                fbanks_row_h[nonZeroNum++] = k;
            }
        }

        fbanks_col_h[i + 1] = nonZeroNum;
    }

    fbanks_col_h[NFILTER + 1] = nonZeroNum;

    float* fbanks_val_d;
    int* fbanks_col_d;
    int* fbanks_row_d;

    float* filterBanks_d;   // filter banks output

    hipMalloc((void**)&fbanks_val_d, nonZeroNum * sizeof(float));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipHostMalloc failed!: %s\n", hipGetErrorString(cudaStatus));

        return 1;
    }

    hipMalloc((void**)&fbanks_col_d, (NFILTER + 2) * sizeof(int));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipHostMalloc failed!: %s\n", hipGetErrorString(cudaStatus));

        return 1;
    }

    hipMalloc((void**)&fbanks_row_d, nonZeroNum * sizeof(int));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipHostMalloc failed!: %s\n", hipGetErrorString(cudaStatus));

        return 1;
    }

    cudaStatus = hipMalloc((void**)&filterBanks_d, numFrames * NFILTER * sizeof(float));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!: %s\n", hipGetErrorString(cudaStatus));

        return 1;
    }

    cudaStatus = hipMemcpy(fbanks_val_d, fbanks_val_h, nonZeroNum * sizeof(float), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed! signal_d: %s\n", hipGetErrorString(cudaStatus));

        return 1;
    }

    cudaStatus = hipMemcpy(fbanks_col_d, fbanks_col_h, (NFILTER + 2) * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed! signal_d: %s\n", hipGetErrorString(cudaStatus));

        return 1;
    }

    cudaStatus = hipMemcpy(fbanks_row_d, fbanks_row_h, nonZeroNum * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed! signal_d: %s\n", hipGetErrorString(cudaStatus));

        return 1;
    }

    //----------------------------------//
    //              MFCCs               //
    //----------------------------------//

    float* mfcc_d;
    cudaStatus = hipMalloc((void**)&mfcc_d, numFrames * NUM_CEPS * sizeof(float));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!: %s\n", hipGetErrorString(cudaStatus));
    
        return 1;
    }
    
    //----------------------------------//
    //         Mean Normalization       //
    //----------------------------------//

    float* mfcc_h;

    cudaStatus = hipHostMalloc((void**)&mfcc_h, numFrames * NUM_CEPS * sizeof(float));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipHostMalloc failed!: %s\n", hipGetErrorString(cudaStatus));

        return 1;
    }

    int p2 = pow(2, ceil(log(numFrames / 2) / log(2)));

    float* meanNorm_d;
    cudaStatus = hipMalloc((void**)&meanNorm_d, p2 * NUM_CEPS * sizeof(float));
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMalloc failed!: %s\n", hipGetErrorString(cudaStatus));

        return 1;
    }

    //----------------------------------//
    //          Execute Kernels         //
    //----------------------------------//

    float ms = 0.0;

    cudaStatus = hipEventRecord(startEvent);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipEventRecord: startEvent, failed!");

        return 1;
    }

    cudaStatus = hipMemcpy(signal_d, signal_h, signalLen * sizeof(short), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed! signal_d: %s\n", hipGetErrorString(cudaStatus));

        return 1;
    }

    PreEmphasis <<< ceil(double(signalLen) / 1024.0), 1024 >>> (PRE_EMPHASIS, signalLen, signal_d, emphasised_d);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "PreEmphasis launch failed: %s\n", hipGetErrorString(cudaStatus));

        return 1;
    }

    PowerFFT <<< numFrames, NFFT / 2, 512 * 3 * sizeof(float2) >>> (BIT_LENGTH, frameLength, frameStep, signalLen, emphasised_d, powFrames_d);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "PowerFFT launch failed: %s\n", hipGetErrorString(cudaStatus));

        return 1;
    }

    MelFilterBank_Sparse <<< numFrames, NFILTER >>> (numFrames, fftResultLen, NFILTER, powFrames_d, filterBanks_d, fbanks_val_d, fbanks_col_d, fbanks_row_d);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "MelFilterBank_Sparse launch failed: %s\n", hipGetErrorString(cudaStatus));

        return 1;
    }

    DCT <<< numFrames, NFILTER >>> (numFrames, NFILTER, NUM_CEPS, CEP_LIFTER, filterBanks_d, mfcc_d);

    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "DCT launch failed: %s\n", hipGetErrorString(cudaStatus));

        return 1;
    }

    if (p2 > 1024)
    {
        MeanNorm_global << < NUM_CEPS, p2 / 4 >> > (numFrames, NUM_CEPS, mfcc_d, meanNorm_d);
        MeanNorm_global << < NUM_CEPS, p2 / 4 >> > (numFrames, NUM_CEPS, mfcc_d, meanNorm_d);
        MeanNorm_global << < NUM_CEPS, p2 / 4 >> > (numFrames, NUM_CEPS, mfcc_d, meanNorm_d);
        MeanNorm_global << < NUM_CEPS, p2 / 4 >> > (numFrames, NUM_CEPS, mfcc_d, meanNorm_d);
    }
    else if (p2 > 512)
    {
        MeanNorm_global <<< NUM_CEPS, p2 >>> (numFrames, NUM_CEPS, mfcc_d, meanNorm_d);
    }
    else
    {
        MeanNorm <<< NUM_CEPS, p2, p2 * sizeof(float) >>> (numFrames, NUM_CEPS, mfcc_d);
    }
    
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "MeanNorm launch failed: %s\n", hipGetErrorString(cudaStatus));
    
        return 1;
    }

    cudaStatus = hipMemcpy(mfcc_h, mfcc_d, numFrames * NUM_CEPS * sizeof(float), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipMemcpy failed! signal_d: %s\n", hipGetErrorString(cudaStatus));

        return 1;
    }

    // stop to record event. 
    cudaStatus = hipEventRecord(stopEvent);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipEventRecord: stopEvent, failed!");

        return 1;
    }

    cudaStatus = hipEventSynchronize(stopEvent);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipEventSynchronize: stopEvent, failed!");

        return 1;
    }

    cudaStatus = hipEventElapsedTime(&ms, startEvent, stopEvent);
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipEventElapsedTime failed!");

        return 1;
    }

    printf("Serial Time:\t%f\n", ms * 1e+6);

    PrintArray(NUM_CEPS, mfcc_h + (numFrames - 1) * NUM_CEPS);

    //----------------------------------//
    //              Cleaning            //
    //----------------------------------//

    // free host memory
    hipHostFree(signal_h);

    hipHostFree(fbanks_val_h);
    hipHostFree(fbanks_col_h);
    hipHostFree(fbanks_row_h);

    hipHostFree(mfcc_h);

    // free device memory
    hipFree(signal_d);
    hipFree(emphasised_d);

    hipFree(powFrames_d);

    hipFree(filterBanks_d);

    hipFree(fbanks_val_d);
    hipFree(fbanks_col_d);
    hipFree(fbanks_row_d);

    hipFree(meanNorm_d);

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess)
    {
        fprintf(stderr, "hipDeviceReset failed!");

        return 1;
    }

    return 0;
}
